#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <algorithm>    // std::swap

#include "ImageWriter.h"
#include "helper.h"
using namespace std;

int nRows = 0; int nCols = 0; int nSheets = 0;

__global__ void Max_Kernel( unsigned char *d_MaxData, unsigned char * d_data, float* d_WeightedSum, float* d_maxWeightedSum, int nSheets)
{
	int SheetsCount = 0; int Max = 0;
	/*
	 * blockDim.x*blockIdx.x+threadIdx.x
	 * is in row major, but we need to match up with the RAW data, which is column major
	 * blockDim is the nCols, gridDim is the nRows
	*/
	int threadNum = threadIdx.x*gridDim.x+blockIdx.x; // make it to colume major
	d_WeightedSum[threadNum] = 0.0;
	while( SheetsCount < nSheets )
	{
		int CurSheetPos = SheetsCount*gridDim.x*blockDim.x +
											blockDim.x*blockIdx.x+threadIdx.x;
											// follow the original logic because it is column major already

		if( Max < (unsigned int)d_data[CurSheetPos] ) {	Max = (unsigned int)d_data[CurSheetPos];	};

		d_WeightedSum[threadNum] +=  ( (nSheets - SheetsCount + 1)/(nSheets * 1.0) )  * d_data[CurSheetPos];
		SheetsCount++;
	}
	d_MaxData[threadNum] = Max;
	atomicMax( d_maxWeightedSum,  d_WeightedSum[threadNum] ); // cuda API function to prevent race condition
}

__global__ void Sum_Kernel( float* d_WeightedSum, unsigned char* d_FinalWeightedSum, float* d_maxWeightedSum )
{
	int threadNum = threadIdx.x*gridDim.x+blockIdx.x;
	float myMax = d_WeightedSum[threadNum];
	float globalMax = *d_maxWeightedSum;
	unsigned char result = (unsigned char)( ( myMax/globalMax )*255.0 );
	//printf("Thread %d: myMax is %f, globalMax is %f, storing %d\n", threadNum, myMax, globalMax,result);
	d_FinalWeightedSum[threadNum] = result;
}

/*
 * get the voxeldata's position after the nCols, nRows, nSheets changed.
*/
int OldData_CurPos_Handler( int projection, int CurRow, int CurCol, int CurSheet )
{
	int SheetSize = 0; // SheetSize of voxeldata, not projection's SheetSize!!!!
	switch( projection )
	{
		case 2: SheetSize = nRows * nCols;
						return ( nSheets - 1 - CurSheet ) * SheetSize + ( nCols - 1 - CurCol ) * nRows + CurRow;
		case 3: SheetSize = nSheets * nCols;
						return CurCol * SheetSize + ( nSheets - CurSheet - 1 ) * nRows + CurRow;
		case 4: SheetSize = nSheets * nCols;
						return (nCols - 1 - CurCol) * SheetSize + CurSheet * nRows + CurRow;
		case 5: SheetSize = nSheets * nCols;
						return CurRow *SheetSize + CurCol * nSheets + CurSheet;
		case 6: SheetSize = nSheets * nCols;
						return (nRows - 1 - CurRow) *SheetSize + CurCol * nSheets + CurSheet; // this is right
	}
	return -1;
}

/*
 * if the projection isn't 1, do the data swaping based on what projection it is
*/
unsigned char* ProjectionCaseHandler( int projection, unsigned char* h_data )
{
	switch( projection ) // change the value of nRows, nCols, nSheets depends on the projection
	{
		case 3:
		case 4: swap(nCols, nSheets); break;
		case 5:
		case 6: swap(nRows, nSheets); break;
	}
	int TotalImageSize = nRows * nCols * nSheets;
	//int SheetSize = nRows * nCols;
	unsigned char* h_NewData = new unsigned char[ TotalImageSize ];
	int NewData_CurPos = 0;
	for( int CurSheet = 0; CurSheet < nSheets; CurSheet++ )
	{
		for( int CurCol = 0; CurCol < nCols; CurCol++ )
		{
			for( int CurRow = 0; CurRow < nRows; CurRow++ )
			{
				int OldData_CurPos = OldData_CurPos_Handler( projection, CurRow, CurCol, CurSheet );
				h_NewData[ NewData_CurPos ] = h_data[ OldData_CurPos ];
				NewData_CurPos++;
			}
		}
	}
	delete h_data;
	return h_NewData;
}

int main(int argc, char** argv)
{
	if( argc < 7 )
	{
		cout<<"not enough arguments."<<endl;
		return -1;
	}
	nRows = atoi(argv[1]); nCols = atoi(argv[2]); nSheets = atoi(argv[3]); int projection = atoi(argv[5]); string filename = argv[6];
	int size = nRows * nCols * nSheets;
	unsigned char* h_data;
	unsigned char* h_MaxData = new unsigned char[nRows * nCols];
	unsigned char* h_FinalWeighedSum;
	unsigned char* d_data;
	unsigned char* d_MaxData;
	unsigned char* d_FinalWeightedSum;
	float* 				 d_WeightedSum;
	float*         d_maxWeightedSum;
	h_data = readfile( argv[4], size ); //read the raw file and returns pointer points to the raw h_data

	/*
	 * if projection isn't 1, change the RAW data to that projection
	*/
	if( projection > 1 )	{ 	h_data = ProjectionCaseHandler( projection, h_data ); 	}

	/*
	 * allocate memory on GPU and launch the Max_Kernelkernel
	*/
	cudaCheck( hipMalloc( (void **)&d_MaxData, sizeof(unsigned char) * nRows * nCols ) ); // malloc the memory for the max data
	cudaCheck( hipMalloc( (void **)&d_data, sizeof(unsigned char)*size ) );
	cudaCheck( hipMalloc( (void **)&d_WeightedSum, sizeof(float) * nRows * nCols ) ); // malloc the memory for the sum value, but not the final sum data
	cudaCheck( hipMalloc( (void **)&d_maxWeightedSum, sizeof(float) ) ); // store the maxvalue in graphics card since we have to get this data from kernel.
	hipMemcpy( d_data, h_data, sizeof(unsigned char)*size, hipMemcpyHostToDevice );
	Max_Kernel<<<nCols, nRows>>>(d_MaxData, d_data, d_WeightedSum, d_maxWeightedSum, nSheets);
	cudaCheck( hipDeviceSynchronize() );

  /*
	 * copy data back to CPU and output the MAX image out
	*/
  cudaCheck( hipMemcpy( h_MaxData, d_MaxData, nRows * nCols * sizeof(unsigned char), hipMemcpyDeviceToHost ) );
	string name = filename+"MAX.png";
	writeTheFile(name, nCols, nRows, h_MaxData);

	/*
	 * running the sum kernel and output the sum image out
	*/
	h_FinalWeighedSum = h_MaxData; d_FinalWeightedSum = d_MaxData; // reuse the same area of d_MaxData since d_MaxData isn't needed anymore
	Sum_Kernel<<<nCols, nRows>>>( d_WeightedSum, d_FinalWeightedSum, d_maxWeightedSum );
	cudaCheck( hipDeviceSynchronize() );
	cudaCheck( hipMemcpy( h_FinalWeighedSum, d_FinalWeightedSum, nRows * nCols * sizeof(unsigned char), hipMemcpyDeviceToHost ) );
	name = filename+"SUM.png";
	writeTheFile(name, nCols, nRows, h_MaxData);

	/*
	 * deallocate memory
	*/
	cudaCheck( hipFree( d_FinalWeightedSum ) );
	cudaCheck( hipFree( d_data ) );
	cudaCheck( hipFree( d_WeightedSum ) ) ;
	cudaCheck( hipFree( d_maxWeightedSum) );
	delete h_MaxData;
	delete h_data;
	return 1;
}
